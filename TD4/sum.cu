
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <math.h>
#include <cfloat>
#include <climits>
#include <thread>
#include <numeric>
#include <iterator>
#include <vector>
#include <atomic>
#include <cmath>
#include <mutex>

//------------------------------------------------

double Sum(double* arr, size_t N) {
    double result = 0.;
    for (size_t i = 0; i < N; ++i) {
        result += arr[i];
    }
    return result;
}

//-------------------------------------------------

void SumMapThread(double* arr, size_t begin, size_t end, double& result) {
    while (begin != end)
        result += arr[begin++];
}

void SumParallel(double* result, size_t N, size_t num_threads) {
    size_t block_size = N / num_threads;
    std::vector<double> results(num_threads, 0.0);

    std::vector<std::thread> workers(num_threads - 1);
    for (size_t i = 0; i < num_threads - 1; i++)
        workers[i] = std::thread(&SumMapThread,
                                result,
                                (i*block_size),
                                ((i+1)*block_size),
                                std::ref(results[i]));
    SumMapThread(result, (num_threads-1)*block_size, N, results[num_threads - 1]);

    for (auto& t : workers) t.join();
    result[0] = std::accumulate(results.begin(), results.end(), 0.0);
}

__global__ void PartialSumGPUAux(double* arr, double* partial_sums, size_t N, size_t chunk_size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t begin = chunk_size * index;
    size_t end = chunk_size * (index + 1);
    double sum = 0.;

    if (end > N)
        end = N;

    for (size_t i = begin; i < end; ++i)
        sum += arr[i];

    partial_sums[index] = sum;
}

/**
 * @brief Computes the sum of the array
 * @param arr - the pointer to the beginning of an array
 * @param N - the length of the array
 */
double SumGPU(double* arr, size_t N) {
    // implement sum of the array on GPU using cuda
    const size_t BLOCKS_NUM = 64;
    const size_t THREADS_PER_BLOCK = 256;
    const size_t TOTAL_THREADS = BLOCKS_NUM  * THREADS_PER_BLOCK;

    // moving the data to device
    double* result = new double[TOTAL_THREADS];
    double* arrGPU;
    double* partialResGPU;
    hipMalloc(&arrGPU, N * sizeof(double));
    hipMalloc(&partialResGPU, TOTAL_THREADS * sizeof(double));
    hipMemcpy(arrGPU, arr, N * sizeof(double), hipMemcpyHostToDevice);

    // computing on GPU
    size_t chunk_size = (N + TOTAL_THREADS + 1) / TOTAL_THREADS;
    PartialSumGPUAux<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(arrGPU, partialResGPU, N, chunk_size);
    hipDeviceSynchronize();

    // copying the result back
    hipMemcpy(result, partialResGPU, TOTAL_THREADS * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(arrGPU);
    hipFree(partialResGPU);

    SumParallel(result, TOTAL_THREADS, 8);
    double res = result[0];
    delete[] result;

    return res;
}

//---------------------------------------------------

int main(int argc, char* argv[]) {
    // setting the random seed to get the same result each time
    srand(42);

    // taking as input, which algo to run
    int alg_ind = std::stoi(argv[1]);

    // Generating data
    size_t N = 1 << 26;
    double* arr = (double*) malloc(N * sizeof(double));
    for (size_t i = 0; i < N; ++i) {
          arr[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
    }

    double result = 0.;
    auto start = std::chrono::steady_clock::now();
    switch (alg_ind) {
        case 0:
            result = Sum(arr, N);
            break;
        case 1:
            result = SumGPU(arr, N);
            break;
    }
    auto finish = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count();
    std::cout << "Elapsed time: " << elapsed << std::endl;
    std::cout << "Total result: " << result << std::endl;

    delete[] arr;
    return 0;
}

