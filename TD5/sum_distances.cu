
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <cmath>

__device__
double DistKer(double* p, double* q, size_t dim) {
    double result = 0;
    for (size_t i = 0; i < dim; ++i) {
        result += (p[i] - q[i]) * (p[i] - q[i]);
    }
    return std::sqrt(result);
}

double Dist(double* p, double* q, size_t dim) {
    double result = 0;
    for (size_t i = 0; i < dim; ++i) {
        result += (p[i] - q[i]) * (p[i] - q[i]);
    }
    return std::sqrt(result);
}

//------------------------------------------------

double SumDistances(double* arr, size_t dim, size_t N) {
    double result = 0.;
    for (size_t i = 0; i < N; ++i) {
        double* p = arr + i * dim;
        for (size_t j = i + 1; j < N; ++j) {
            result += Dist(p, arr + j * dim, dim);
        }
    }
    return result;
}

//-------------------------------------------------

__global__
void SumDistancesGPUAux(double* arr, size_t dim , size_t N, double* results_gpu) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) {
        return;
    }
    double result = 0.;
    double* p = arr + index * dim;
    for (size_t i = index + 1; i < N; ++i) {
        result += DistKer(p, arr + i * dim, dim);
    }
    results_gpu[index] = result;
}

double SumDistancesGPU(double* arr, size_t dim, size_t N) {
    const size_t THREADS_PER_BLOCK = 256;

    // moving the input to the device
    double* arr_device;
    hipMalloc(&arr_device, N * dim * sizeof(double));
    hipMemcpy(arr_device, arr, N * dim * sizeof(double), hipMemcpyHostToDevice);

    // allocating memory for the output
    double* results_gpu;
    hipMalloc(&results_gpu, N * sizeof(double));

    // running calculation on GPU
    size_t blocks_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    SumDistancesGPUAux<<<blocks_num, THREADS_PER_BLOCK>>>(arr_device, dim, N, results_gpu);
    hipDeviceSynchronize();

    // Computing the final result
    double* results_cpu = (double*) malloc(N * sizeof(double));
    hipMemcpy(results_cpu, results_gpu, N * sizeof(double), hipMemcpyDeviceToHost);

    double final_result = 0;
    for (size_t i = 0; i < N; ++i) {
        final_result += results_cpu[i];
    }

    // Freeing the memory
    hipFree(arr_device);
    hipFree(results_gpu);
    free(results_cpu);
    return final_result;
}

//---------------------------------------------------

__global__ void SumDistancesGPUAux2(double* arr, size_t dim , size_t N, double* results_gpu) {
    const size_t tid = threadIdx.x;
    const size_t index = blockIdx.x * blockDim.x + tid;
    if (index >= N) return;

    __shared__ double* p_curr[256];
    double* p = arr + index * dim;
    p_curr[tid] = p;
    __syncthreads();
    double result = 0.;

    for (size_t i = tid + 1; i < 256; ++i)
        result += DistKer(p, *(p_curr + i), dim);

    __shared__ double* p_next[256];
    for (size_t i = 0; i < N / 256; ++i) {
        size_t next_index = index + 256 * (i + 1);
        if (next_index >= N) break;

        double* p_ = arr + next_index * dim;
        p_next[tid] = p_;
        __syncthreads();

        for (size_t j = 0; j < 256; ++j)
            result += DistKer(p, *(p_next + j), dim);
    }

    results_gpu[index] = result;
}

double SumDistancesGPU2(double* arr, size_t dim, size_t N) {
    const size_t THREADS_PER_BLOCK = 256;
    const size_t blocks_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Allocating memory on the device
    double* arr_device;
    double* results_gpu;
    double* results_cpu = (double*) malloc(N * sizeof(double));
    hipMalloc(&arr_device, N * dim * sizeof(double));
    hipMalloc(&results_gpu, N * sizeof(double));

    // moving the input to the device
    hipMemcpy(arr_device, arr, N * dim * sizeof(double), hipMemcpyHostToDevice);

    // running calculation on GPU
    SumDistancesGPUAux2<<<blocks_num, THREADS_PER_BLOCK>>>(arr_device, dim, N, results_gpu);
    hipDeviceSynchronize();

    // Computing the final result
    hipMemcpy(results_cpu, results_gpu, N * sizeof(double), hipMemcpyDeviceToHost);

    double final_result = 0;
    for (size_t i = 0; i < N; ++i)
        final_result += results_cpu[i];

    // Freeing the memory
    hipFree(arr_device);
    hipFree(results_gpu);
    free(results_cpu);
    return final_result;
}

//---------------------------------------------------

int main(int argc, char* argv[]) {
    // setting the random seed to get the same result each time
    srand(42);

    // taking as input, which algo to run
    int alg_ind = std::stoi(argv[1]);

    // Generating data
    size_t N = 6400;
    size_t dim = 3;
    double* arr = (double*) malloc(N * dim * sizeof(double));
    for (size_t i = 0; i < dim * N; ++i) {
          arr[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
    }

    // Computing on CPU
    double result = 0.;
    auto start = std::chrono::steady_clock::now();
    switch (alg_ind) {
        case 0:
            result = SumDistances(arr, dim, N);
            break;
        case 1:
            result = SumDistancesGPU(arr, dim, N);
            break;
        case 2:
            result = SumDistancesGPU2(arr, dim, N);
            break;
    }
    auto finish = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count();
    std::cout << "Elapsed time: " << elapsed << std::endl;
    std::cout << "Total result: " << result << std::endl;

    free(arr);
    return 0;
}
