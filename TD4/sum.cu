
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <math.h>

//------------------------------------------------

double Sum(double* arr, size_t N) {
    double result = 0.;
    for (size_t i = 0; i < N; ++i) {
        result += arr[i];
    }
    return result;
}

//-------------------------------------------------


/**
 * @brief Computes the sum of the array
 * @param arr - the pointer to the beginning of an array
 * @param N - the length of the array
 */
double SumGPU(double* arr, size_t N) {
    return 0.;
}

//---------------------------------------------------

int main(int argc, char* argv[]) {
    // setting the random seed to get the same result each time
    srand(42);

    // taking as input, which algo to run
    int alg_ind = std::stoi(argv[1]);

    // Generating data
    size_t N = 1 << 26;
    double* arr = (double*) malloc(N * sizeof(double));
    for (size_t i = 0; i < N; ++i) {
          arr[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
    }
 
    double result = 0.;
    auto start = std::chrono::steady_clock::now();
    switch (alg_ind) {
        case 0: 
            result = Sum(arr, N);
            break;
        case 1:
            result = SumGPU(arr, N);
            break;
    }
    auto finish = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count(); 
    std::cout << "Elapsed time: " << elapsed << std::endl;
    std::cout << "Total result: " << result << std::endl;
    
    delete[] arr;
    return 0;
}
